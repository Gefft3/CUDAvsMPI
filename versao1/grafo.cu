#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <set>
#include <map>
#include <chrono>

using namespace std;
using namespace chrono;

class Graph {
public:
    vector<pair<int, int>> arestas;
    vector<set<int>> vizinhos;
    set<int> vertices;

    Graph(vector<pair<int, int>> edgeList) {
        for (auto edge : edgeList) {
            int maxVertex = max(edge.first, edge.second);
            if (vizinhos.size() <= maxVertex) {
                vizinhos.resize(maxVertex + 1);
            }
            arestas.push_back(edge);
            vertices.insert(edge.first);
            vertices.insert(edge.second);
            vizinhos[edge.first].insert(edge.second);
            vizinhos[edge.second].insert(edge.first);
        }
    }

    vector<int> getNeighbours(int vertex) {
        vector<int> neighbours(vizinhos[vertex].begin(), vizinhos[vertex].end());
        return neighbours;
    }

    void release() {
        arestas.clear();
        vertices.clear();
        vizinhos.clear();
    }
};

// Função para carregar dataset e renomear vértices
vector<pair<int, int>> rename(const string& dataset) {
    ifstream inputFile(dataset);
    map<int, int> nodeMap;
    vector<pair<int, int>> edges;
    int nodeCounter = 0;

    if (!inputFile.is_open()) {
        cerr << "Erro ao abrir arquivo: " << dataset << endl;
        exit(1);
    }

    int u, v;
    while (inputFile >> u >> v) {
        if (nodeMap.find(u) == nodeMap.end()) {
            nodeMap[u] = nodeCounter++;
        }
        if (nodeMap.find(v) == nodeMap.end()) {
            nodeMap[v] = nodeCounter++;
        }
        edges.emplace_back(nodeMap[u], nodeMap[v]);
    }
    inputFile.close();
    return edges;
}

// Kernel CUDA para contagem de cliques
__global__ void contagem_cliques_kernel(int* d_vizinhos, int* d_offsets, int numVertices, int k, int* d_cliqueCount, int* d_cliqueBuffer, int* d_workQueue, int* d_workQueueIndex) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int queueIndex = atomicAdd(d_workQueueIndex, 1);

    // Se não houver mais trabalho na fila, a thread deve retornar
    if (queueIndex >= numVertices) return;

    if (tid < numVertices){

    // Consumir um vértice da fila global de trabalho
    int vertex = d_workQueue[queueIndex];

    // Inicializar o buffer de clique
    int cliqueStart = 0;
    d_cliqueBuffer[cliqueStart] = vertex;
    int cliqueSize = 1;

    // Contagem de cliques de tamanho k
    for (int cliqueLevel = 1; cliqueLevel < k; cliqueLevel++) {
        int lastVertex = d_cliqueBuffer[cliqueStart + cliqueSize - 1];
        int start = d_offsets[lastVertex];
        int end = d_offsets[lastVertex + 1];
        bool expanded = false;

        for (int i = start; i < end; i++) {
            int vizinho = d_vizinhos[i];
            bool isClique = true;

            // Verifique conectividade com todos os vértices no clique atual
            for (int j = 0; j < cliqueSize; j++) {
                int cliqueVertex = d_cliqueBuffer[cliqueStart + j];
                int neighborStart = d_offsets[cliqueVertex];
                int neighborEnd = d_offsets[cliqueVertex + 1];
                bool found = false;

                for (int n = neighborStart; n < neighborEnd; n++) {
                    if (d_vizinhos[n] == vizinho) {
                        found = true;
                        break;
                    }
                }
                if (!found) {
                    isClique = false;
                    break;
                }
            }

            if (isClique) {
                d_cliqueBuffer[cliqueStart + cliqueSize] = vizinho;
                cliqueSize++;
                expanded = true;

                if (cliqueSize == k) {
                    atomicAdd(d_cliqueCount, 1);
                    break;
                }
            }
        }
        if (!expanded) break;
    }
}
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        cerr << "Uso: " << argv[0] << " <dataset> <k-clique>" << endl;
        return 1;
    }

    string dataset = argv[1];
    int k_clique = atoi(argv[2]);

    // Carregar o grafo
    vector<pair<int, int>> edges = rename(dataset);
    Graph g(edges);
    int numVertices = g.vertices.size();

    // Preparar dados para GPU
    vector<int> vizinhosFlat;
    vector<int> offsets(numVertices + 1, 0);
    for (int v = 0; v < numVertices; v++) {
        vector<int> vizinhos = g.getNeighbours(v);
        offsets[v + 1] = offsets[v] + vizinhos.size();
        vizinhosFlat.insert(vizinhosFlat.end(), vizinhos.begin(), vizinhos.end());
    }

    // Fila global de trabalho com todos os vértices
    vector<int> workQueue(numVertices);
    for (int i = 0; i < numVertices; i++) {
        workQueue[i] = i;
    }

    // Alocar memória na GPU
    int* d_vizinhos, *d_offsets, *d_cliqueCount, *d_cliqueBuffer, *d_workQueue, *d_workQueueIndex;
    int cliqueBufferSize = numVertices * k_clique;
    hipMalloc(&d_vizinhos, vizinhosFlat.size() * sizeof(int));
    hipMalloc(&d_offsets, offsets.size() * sizeof(int));
    hipMalloc(&d_cliqueCount, sizeof(int));
    hipMalloc(&d_cliqueBuffer, cliqueBufferSize * sizeof(int));
    hipMalloc(&d_workQueue, workQueue.size() * sizeof(int));
    hipMalloc(&d_workQueueIndex, sizeof(int));

    // Copiar dados para a GPU
    hipMemcpy(d_vizinhos, vizinhosFlat.data(), vizinhosFlat.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, offsets.data(), offsets.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_workQueue, workQueue.data(), workQueue.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_cliqueCount, 0, sizeof(int));
    hipMemset(d_workQueueIndex, 0, sizeof(int));

    // Configurar kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numVertices + threadsPerBlock - 1) / threadsPerBlock;

    auto start = high_resolution_clock::now();
    contagem_cliques_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_vizinhos, d_offsets, numVertices, k_clique, d_cliqueCount, d_cliqueBuffer, d_workQueue, d_workQueueIndex);
    hipDeviceSynchronize();

    // Verificar erros
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        cerr << "Erro no kernel: " << hipGetErrorString(err) << endl;
        return 1;
    }

    auto end = high_resolution_clock::now();

    // Recuperar resultado
    int cliqueCount;
    hipMemcpy(&cliqueCount, d_cliqueCount, sizeof(int), hipMemcpyDeviceToHost);

    // Exibir resultados
    duration<double> duration = end - start;
    cout << "Número de cliques: " << cliqueCount << endl;
    cout << "Tempo de execução: " << duration.count() << " segundos" << endl;

    // Liberar memória
    hipFree(d_vizinhos);
    hipFree(d_offsets);
    hipFree(d_cliqueCount);
    hipFree(d_cliqueBuffer);
    hipFree(d_workQueue);
    hipFree(d_workQueueIndex);

    g.release();
    return 0;
}
